/**
 * @file   : constant_eg.cu
 * @brief  : Examples of using constant memory for CUDA 
 * @details : constant memory for CUDA examples
 *  
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20170103      
 * @ref    : http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#device-memory-specifiers
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * nvcc constant_eg.cu -o constant_eg
 * 
 * */

#include <hip/hip_runtime.h>
#include <iostream>   

__constant__ float constData_global[256]; 
__device__ float devData;  
__device__ float* devPointer; 


int main(int argc, char* argv[]) {
	float data_main[256]; 

	/* "boilerplate" test values */ 
	for (int idx=0; idx<256; idx++) { 
		data_main[idx] = ((float) idx+1);  
	}
	
	hipMemcpyToSymbol(HIP_SYMBOL(constData_global), data_main, sizeof(data_main)); 
	
	float data_main1[256]; 
	for (int idx=0; idx < 256; idx++) { std::cout << data_main1[idx] << " "; }

	hipMemcpyFromSymbol(data_main1, HIP_SYMBOL(constData_global), sizeof(data_main1) );
	/* sanity check */ 
	for (int idx=0; idx < 256; idx++) { std::cout << data_main1[idx] << " "; }
	
//	__constant__ float constData_main[256]; // error:  a "__constant__" 
	// variable declaration is not allowed inside a function body

//	__device__ float devData; // error: a "__device__" variable declaration is not allowed inside a function body
	float value = 3.14; 
	hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)); 
	
	float *ptr; 
	hipMalloc(&ptr, 256*sizeof(float)); 
	hipMemcpyToSymbol(HIP_SYMBOL(devPointer), &ptr, sizeof(ptr));  
	
	

}
