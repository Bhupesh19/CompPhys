#include "hip/hip_runtime.h"
/**
 * @file   : sysparam.cu
 * @brief  : Physical parameters of the system separate implementation file, in CUDA C++11/14, on device GPU 
 * @details : as a struct, have total energy E, total magnetization M, temperature of entire system T (in energy units)  
 * 
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20180103    
 * @ref    : M. Hjorth-Jensen, Computational Physics, University of Oslo (2015)
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * g++ main.cpp ./structs/structs.cpp -o main
 * 
 * */
#include "./sysparam.h"

/* =============== device GPU structs =============== */ 

// getting functions
/** @fn get_by_DeltaE 
 * @details given DeltaE (\Delta E), DeltaE = -8J, -4J,...8J, we want to get the 
 * transition probability from std::unique_ptr transprob (but transprob indexed by 
 * 0,1,...(17-1)
 * */
float TransProb::get_by_DeltaE(const int DeltaE) {
		return transProb[DeltaE+8]; 
} 

  
/* struct of structs
 * I chose this design because 
 * 1. I want a struct such that its data members is identified by pseudo-mathematical symbols, 
 * i.e. a data member E should denote total Energy  
 * 1.a. I want this struct to be "empty" in that it has no custom constructors/destructors, 
 * so that it can be used by both host CPU and device GPU, and can be qualified with __constant__, so to be 
 * put in constant memory 
 * 2. I want a struct of these structs so to automate its construction/destruction: 
 * hipMallocManaged, hipFree, and use smart pointers for it
 * */

// default constructor
/** @fn Sysparam_ptr()
 * @brief default constructor for struct Sysparam_ptr 
 * @details set all E,M,T parameters to 0
 * */
Sysparam_ptr::Sysparam_ptr() { 
	std::unique_ptr<Sysparam, del_Sysparam_struct> d_sysparams_in(nullptr, del_Sysparam_struct() ); 
	hipMallocManaged((void **) &d_sysparams_in, 1 * sizeof(Sysparam)) ;
	d_sysparams = std::move(d_sysparams_in);  
	
	d_sysparams->E = 0.f; d_sysparams->M = 0.f; d_sysparams->T = 0.f;}  

// constructors
/** @fn Sysparam_ptr(const float,const float,const float)
 * @brief constructor for struct Sysparam_ptr 
 * */
Sysparam_ptr::Sysparam_ptr(const float E, const float M, const float T) {  
	std::unique_ptr<Sysparam, del_Sysparam_struct> d_sysparams_in(nullptr, del_Sysparam_struct() ); 
	hipMallocManaged((void **) &d_sysparams_in, 1 * sizeof(Sysparam)) ;
	d_sysparams = std::move(d_sysparams_in);  

	d_sysparams->E = E; d_sysparams->M = M; d_sysparams->T = T; }  

/** @fn Sysparam_ptr(const float)
 * @brief constructor for struct Sysparam_ptr, when only given the system temperature (initially)
 * */
Sysparam_ptr::Sysparam_ptr(const float T) { 
	std::unique_ptr<Sysparam, del_Sysparam_struct> d_sysparams_in(nullptr, del_Sysparam_struct() ); 
	hipMallocManaged((void **) &d_sysparams_in, 1 * sizeof(Sysparam)) ;
	d_sysparams = std::move(d_sysparams_in);  

	d_sysparams->E = 0.f; d_sysparams->M = 0.f; d_sysparams->T = T; }  

// move constructor; necessitated by unique_ptr
Sysparam_ptr::Sysparam_ptr(Sysparam_ptr && old_sysparam_ptr) : 
	d_sysparams { std::move(old_sysparam_ptr.d_sysparams) }  {}     

// operator overload assignment = 
Sysparam_ptr & Sysparam_ptr::operator=(Sysparam_ptr && old_sysparam_ptr) {
	d_sysparams = std::move( old_sysparam_ptr.d_sysparams );
	return *this;
}



// default constructor
Avg_ptr::Avg_ptr() { 
	std::unique_ptr<Avg, del_Avg_struct> d_avgs_in(nullptr, del_Avg_struct() ); 
	hipMallocManaged((void **) &d_avgs_in, 1 * sizeof(Avg)) ;
	d_avgs = std::move(d_avgs_in);  
	
	d_avgs->Eavg = 0.f; d_avgs->Mavg = 0.f; d_avgs->Esq_avg = 0.f; d_avgs->Msq_avg = 0.f; 
	d_avgs->absM_avg = 0.f; d_avgs->M4_avg = 0.f; 
}  

// constructors
Avg_ptr::Avg_ptr(const float Eavg, const float Mavg, const float Esq_avg, const float Msq_avg,
	const float absM_avg, const float M4_avg) { 
	std::unique_ptr<Avg, del_Avg_struct> d_avgs_in(nullptr, del_Avg_struct() ); 
	hipMallocManaged((void **) &d_avgs_in, 1 * sizeof(Avg)) ;
	d_avgs = std::move(d_avgs_in);  
	
	d_avgs->Eavg = Eavg; d_avgs->Mavg = Mavg; d_avgs->Esq_avg = Esq_avg; d_avgs->Msq_avg = Msq_avg; 
	d_avgs->absM_avg = absM_avg; d_avgs->M4_avg = M4_avg; 
}  

// move constructor; necessitated by unique_ptr
Avg_ptr::Avg_ptr(Avg_ptr && old_avg_ptr) :  
	  d_avgs { std::move(old_avg_ptr.d_avgs) } {}

// operator overload assignment = 
Avg_ptr & Avg_ptr::operator=(Avg_ptr && old_avg_ptr)  {
	d_avgs = std::move( old_avg_ptr.d_avgs );
	return *this; 
}


// default constructor
TransProb_ptr::TransProb_ptr() { 
	std::unique_ptr<TransProb, del_TransProb_struct> d_transProb_in(nullptr, del_TransProb_struct() ); 
	hipMallocManaged((void **) &d_transProb_in, 1 * sizeof(TransProb)) ;
	d_transProb = std::move(d_transProb_in);  
}	

// constructors
TransProb_ptr::TransProb_ptr(const float T, const float J) { 
	std::unique_ptr<TransProb, del_TransProb_struct> d_transProb_in(nullptr, del_TransProb_struct() ); 
	hipMallocManaged((void **) &d_transProb_in, 1 * sizeof(TransProb)) ;
	d_transProb = std::move(d_transProb_in);  

	d_transProb->J = J; 
	for (int de = -8; de <= 8; de +=4) {
		(d_transProb->transProb)[de+8] = std::exp(-((float) de)/T); 
	}
	
//	hipMemcpyToSymbol(HIP_SYMBOL(constTransProb), &(this->d_transProb), sizeof(TransProb)*1); 
	
}	

// move constructor; necessitated by unique_ptr
TransProb_ptr::TransProb_ptr(TransProb_ptr && old_transProb_ptr) : 
	d_transProb { std::move(old_transProb_ptr.d_transProb ) }  {}   

// operator overload assignment = 
TransProb_ptr & TransProb_ptr::operator=(TransProb_ptr && old_transProb_ptr)  {
	d_transProb = std::move( old_transProb_ptr.d_transProb );
	return *this; 
}



/* ********** END of device GPU structs ********** */

